#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

struct Startup{
    int seed = time(nullptr);
    int random_range = 100;
    int threads_per_block = 5;
} startup;

/*
 Found on the stack overflow:  https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
 Throws errors if cuda command doesn't return Success
*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct DataSet{
    float* values;
    int  size;
};

inline int sizeOfDataSet(DataSet data){ return sizeof(float)*data.size; }

DataSet generateRandomDataSet(int size){
    DataSet data;
    data.size = size;
    data.values = (float*)malloc(sizeof(float)*data.size);

    for (int i = 0; i < data.size; i++)
        data.values[i] = (float)(rand()%startup.random_range);

    return data;
}

__global__ void DeviceCalculateSMA(float* input, int input_size, float* result, int result_size, int sample_size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < result_size){

        extern __shared__ float cache[];

        /*Load Subset of values into shared memory*/

        int cachedDataSize = sample_size + blockDim.x;

        for (int i = 0; i < cachedDataSize/blockDim.x; i++){
            cache[threadIdx.x+ i*blockDim.x] = input[threadIdx.x+ i*blockDim.x];
        }
        __syncthreads();

        float sum = 0;
        for (int i = 0; i < sample_size; i++)
            sum += cache[(idx+i)%(blockDim.x+sample_size)];
        sum /= sample_size;

        result[idx] = sum;
    }

}

DataSet CalculateSMA(DataSet input, int sample_size){
    DataSet host_result = {(float*)malloc(sizeof(float)*(input.size-sample_size)), input.size-sample_size};

    float* device_input, *device_result;

    gpuErrchk(hipMalloc((void **)&device_input,  sizeOfDataSet(input) ));
    gpuErrchk(hipMalloc((void **)&device_result, sizeOfDataSet(host_result) ));

    gpuErrchk(hipMemcpy(device_input, input.values, sizeOfDataSet(input) , hipMemcpyHostToDevice));

    int threads_needed = host_result.size;
    DeviceCalculateSMA<<<threads_needed/ startup.threads_per_block + 1, startup.threads_per_block, sizeof(float)*(host_result.size+sample_size)>>> (device_input, input.size, device_result, host_result.size, sample_size);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipMemcpy(host_result.values, device_result, sizeOfDataSet(host_result), hipMemcpyDeviceToHost));

    return host_result;
}

void printDataSetI(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.0f,", data.values[i]);
    printf("\n");
}

void printDataSetF(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.1f ", data.values[i]);
    printf("\n");
}


int main(int argc, char** argv){
    srand(startup.seed);

    DataSet data = generateRandomDataSet(100);
    printDataSetI( data );
    DataSet result = CalculateSMA(data, 5);
    printf("\n");
    printDataSetF( result );
}