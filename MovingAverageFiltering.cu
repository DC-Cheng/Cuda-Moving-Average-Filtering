#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

struct Startup{
    int seed = time(nullptr);
    int random_range = 100;
    int threads_per_block = 1;
} startup;

/*
 Found on the stack overflow:  https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
 Throws errors if cuda command doesn't return Success
*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct DataSet{
    float* values;
    int  size;
};

inline int sizeOfDataSet(DataSet data){ return sizeof(float)*data.size; }

DataSet generateRandomDataSet(int size){
    DataSet data;
    data.size = size;
    data.values = (float*)malloc(sizeof(float)*data.size);

    for (int i = 0; i < data.size; i++)
        data.values[i] = (float)(rand()%startup.random_range);

    return data;
}

__global__ void DeviceCalculateSMA_Global(float* input, int input_size, float* result, int result_size, int sample_size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < result_size){

        float sum = 0;
        for (int i = 0; i < sample_size; i++)
            sum += input[idx+i];
        sum /= sample_size;

        result[idx] = sum;
    }
}

__global__ void DeviceCalculateSMA_Shared(float* input, int input_size, float* result, int result_size, int sample_size){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < result_size){

        extern __shared__ float cache[];

        int cachedDataSize = sample_size + blockDim.x;

        /*Copy the data that will be used by the block into shared memory using all threads in the block.*/
        for (int i = 0; i < cachedDataSize/blockDim.x; i++){
            int cacheId = threadIdx.x+ i*blockDim.x;
            if (cacheId < cachedDataSize)
                cache[cacheId] = input[cacheId+blockDim.x * blockIdx.x];
        }
        __syncthreads();

        float sum = 0;
        for (int i = 0; i < sample_size; i++){
            sum += cache[i+threadIdx.x];
        }

        sum /= sample_size;
        result[idx] = sum;
    }

}

DataSet CalculateSMA(DataSet input, int sample_size){
    int result_size = input.size-sample_size+1;
    DataSet host_result = {(float*)malloc(sizeof(float)*(result_size)), result_size};

    float* device_input, *device_result;

    gpuErrchk(hipMalloc((void **)&device_input,  sizeOfDataSet(input) ));
    gpuErrchk(hipMalloc((void **)&device_result, sizeOfDataSet(host_result) ));

    gpuErrchk(hipMemcpy(device_input, input.values, sizeOfDataSet(input) , hipMemcpyHostToDevice));

    int threads_needed = host_result.size;
    //DeviceCalculateSMA_Global<<<threads_needed/ startup.threads_per_block + 1, startup.threads_per_block>>> (device_input, input.size, device_result, host_result.size, sample_size);
    DeviceCalculateSMA_Shared<<<threads_needed/ startup.threads_per_block + 1, startup.threads_per_block, sizeof(float)*(input.size)>>> (device_input, input.size, device_result, host_result.size, sample_size);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipMemcpy(host_result.values, device_result, sizeOfDataSet(host_result), hipMemcpyDeviceToHost));

    return host_result;
}

void printDataSetI(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.0f,", data.values[i]);
    printf("\n");
}

void printDataSetF(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.4f ", data.values[i]);
    printf("\n");
}


int main(int argc, char** argv){
    srand(0);

    DataSet data = generateRandomDataSet(100);
    printDataSetI( data );
    DataSet result = CalculateSMA(data, 16);
    printf("\n");
    printDataSetF( result );
}