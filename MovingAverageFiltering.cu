#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

struct Startup{
    int seed = time(nullptr);
    int random_range = 10;
} startup;

/*
 Found on the stack overflow:  https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
 Throws errors if cuda command doesn't return Success
*/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

struct DataSet{
    float* values;
    int  size;
};

inline int sizeOfDataSet(DataSet data){ return sizeof(float)*data.size; }

DataSet generateRandomDataSet(int size){
    DataSet data;
    data.size = size;
    data.values = (float*)malloc(sizeof(float)*data.size);

    for (int i = 0; i < data.size; i++)
        data.values[i] = (float)(rand()%startup.random_range);

    return data;
}

DataSet cudaCalculateSMA(DataSet input, int sample_size){
    DataSet host_result = {(float*)malloc(sizeof(float)*(input.size-sample_size)), input.size-sample_size};

    float* device_input, device_result;

    gpuErrchk(hipMalloc((void **)&device_input,  sizeOfDataSet(input) ));
    gpuErrchk(hipMalloc((void **)&device_result, sizeOfDataSet(host_result) ));

    gpuErrchk(hipMemcpy(device_input, input.values, sizeOfDataSet(input) , hipMemcpyHostToDevice));


    return host_result;
}

void printDataSet(DataSet data){
    for (int i = 0; i < data.size; i++)
        printf("%.0f ", data.values[i]);
    printf("\n");
}

int main(int argc, char** argv){
    printDataSet( generateRandomDataSet(100) );
}